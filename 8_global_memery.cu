#include "hip/hip_runtime.h"
#include<stdio.h>
#include<iostream>
#include<hip/hip_runtime.h>
#include "./tools/common.cuh"


__device__ int d_x = 1;
__device__ int d_y[2];

__global__ void kernel(void)
{
    d_y[0] += d_x;
    d_y[1] += d_x;

    printf("d_x = %d, d_y[0] = %d, d_y[1] = %d.\n", d_x, d_y[0], d_y[1]);
}



int main(int argc, char **argv)
{
    int devID = 0;
    hipDeviceProp_t deviceProps;
    errorCheck(hipGetDeviceProperties(&deviceProps, devID),__FILE__,__LINE__);
    std::cout << "运行GPU设备:" << deviceProps.name << std::endl;

    int h_y[2] = {10, 20};
    errorCheck(hipMemcpyToSymbol(HIP_SYMBOL(d_y), h_y, sizeof(int) * 2),__FILE__,__LINE__);

    dim3 block(2);
    dim3 grid(2);
    kernel<<<grid, block>>>();
    errorCheck(hipDeviceSynchronize(),__FILE__,__LINE__);
    errorCheck(hipMemcpyFromSymbol(h_y, HIP_SYMBOL(d_y), sizeof(int) * 2),__FILE__,__LINE__);
    printf("h_y[0] = %d, h_y[1] = %d.\n", h_y[0], h_y[1]);

    errorCheck(hipDeviceReset(),__FILE__,__LINE__);

    return 0;
}