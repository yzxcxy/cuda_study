#include <stdio.h>
#include "./tools/common.cuh"

int main(){
    //检测设备中GPU的数量
    int deviceCount=0;
    hipError_t cuda_error;
    cuda_error=errorCheck(hipGetDeviceCount(&deviceCount),__FILE__,__LINE__);

    //判断错误情况
    if(cuda_error!=hipSuccess || deviceCount==0){
        printf("hipGetDeviceCount failed!  Do you have a CUDA-Capable GPU installed?\n");
        exit(-1);
    }else{
        printf("CUDA Device Query...\n");
        printf("There are %d CUDA devices.\n",deviceCount);
    }

    //设置0号设备执行
    int index_device=1;
    cuda_error=errorCheck(hipSetDevice(index_device),__FILE__,__LINE__);
    if(cuda_error!=hipSuccess){
        printf("hipSetDevice failed!  Do you have a CUDA-Capable GPU installed?\n");
        exit(-1);
    }else{
        printf("Set device %d to execute.\n",index_device);
    }

    return 0;
}